#include "hip/hip_runtime.h"
#include "genetic_cuda.h"

__constant__ size_t pop_size;
__constant__ size_t member_size;
__constant__ double crossover_rate;
__constant__ double mutation_rate;

__global__ void genetic_kernel() {
}

void run_genetic_seq(size_t p_size, size_t m_size, double cr_rate,
		double m_rate, unsigned max_iter, double (*fitness_func)(uint8_t*)) {
	hipMemcpyToSymbol(HIP_SYMBOL(pop_size), &p_size, sizeof(size_t));
	hipMemcpyToSymbol(HIP_SYMBOL(member_size), &m_size, sizeof(size_t));
	hipMemcpyToSymbol(HIP_SYMBOL(crossover_rate), &cr_rate, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(mutation_rate), &m_rate, sizeof(double));
	hipDeviceSynchronize();
}
