#include "hip/hip_runtime.h"
#include "genetic_cuda.h"
#include "utils.h"
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

__constant__ static size_t pop_size;
__constant__ static size_t member_size;
__constant__ static double crossover_rate;
__constant__ static double mutation_rate;
__constant__ static unsigned max_iterations;
__constant__ double (*fitness)(uint8_t*);

inline void check_cuda_error(std::string msg) {
        hipError_t err = hipGetLastError();
        if(hipSuccess != err) {
                std::cerr << "\033[1;31mcuda error: " << msg << std::endl;
                std::cerr << "\t" << hipGetErrorString(err) << std::endl;
                exit(1);
        }
}

__device__ static void print_member(uint8_t* member) {
	for(size_t j = 0; j < member_size; j++) {
		printf("%d ", member[j]);
	}
	printf("\n");
}

__device__ static void print_pop(uint8_t* member_arr) {
	for(size_t i = 0; i < pop_size; i++) {
		print_member(&member_arr[i*member_size]);
	}
}

static __device__ uint8_t* get_member(uint8_t* member_array, int i) {
	return &member_array[i * member_size];
}

__device__ void init_population(int idx, uint8_t* population,
	thrust::default_random_engine& rand_gen) {
	thrust::uniform_int_distribution<uint8_t> dist(0, 1);
	for(unsigned i = 0; i < member_size; i++) {
		rand_gen.discard(idx);
		population[member_size * idx + i] = dist(rand_gen);
	}
	#if 0
	if(idx == 0) {
		for(unsigned i = 0; i < member_size; i++)
			population[member_size * idx + i] = 1;
	}
	#endif
}

__device__ void record_fitness(int idx, double* fitness_arr,
	uint8_t* population, double* best_fitnesses) {
	double current_fitness = fitness(get_member(population, idx));
	fitness_arr[idx] = current_fitness;
	if(current_fitness > best_fitnesses[idx])
		best_fitnesses[idx] = current_fitness;
	//TODO: remove the following later
	#if 0
	if(1/current_fitness <= 100)
		printf("fitness: %f\n", 1/current_fitness);
	#endif
}

__device__ static unsigned roulette(double* fitness_arr, double fitness_sum,
	thrust::default_random_engine& rand_gen) {

	rand_gen.discard(threadIdx.x);

	thrust::uniform_real_distribution<double> dist(0.0, fitness_sum);
	//thrust::default_random_engine rand_gen;
	double rand_num = dist(rand_gen);
	
	double partial_sum = 0.0;
	for(unsigned i = 0; i < pop_size; i++) {
		partial_sum += fitness_arr[i];
		if(partial_sum >= rand_num)
			return i;
	}
	return 0.0;
}

__device__ void selection(int idx, uint8_t* population, double* fitness_arr, uint8_t* new_population,
	thrust::default_random_engine& rand_gen, double* best_fitnesses) {
	__shared__ double fitness_sum;
	//__shared__ uint8_t* new_population;
	record_fitness(idx, fitness_arr, population, best_fitnesses);

	#if 0
	if(idx == 0) {
		printf("fitness arr: ");
		for(unsigned i = 0; i < pop_size; i++) {
			printf("%f, ", fitness_arr[i]);
		}
		printf("\n");
	}
	#endif

	__syncthreads();
	prefix_sum(fitness_arr, &fitness_sum, pop_size);

	#if 0
	if(idx == 0) {
		__syncthreads();
		if(idx == 0)
			printf("fitness sum is %f\n", fitness_sum);
		//new_population = new uint8_t[pop_size];
	}
	#endif

	__syncthreads();
	// TODO: parallelize probability calculation
	// TODO: parallelize roulette selection
	if(idx == 0) {
		// prob calc
		double* probs = new double[pop_size]();
		#if 1
		double partial_prob_sum = 0.0;
		for(unsigned i = 0; i < pop_size; i++) {
			probs[i] = partial_prob_sum + fitness_arr[i] / fitness_sum;
			partial_prob_sum += probs[i];
		}

		// roulette
		for(unsigned i = 0; i < pop_size; i++) {
			unsigned selected_idx = roulette(fitness_arr, fitness_sum, rand_gen);
			memcpy(get_member(new_population, i),
				get_member(population, selected_idx), member_size);
		}
		#endif

		// free stuff
		#if 0
		delete[] fitness_arr;
		#endif
		delete[] probs;
		//return new_population;
	}
}

__device__ static void two_point_crossover(int idx, uint8_t* parent1, uint8_t* parent2,
	thrust::default_random_engine& rand_gen) {
	// crossover only with probability of crossover rate
	thrust::uniform_real_distribution<double> crossover_dist(0.0, 1.0);
	rand_gen.discard(idx);
	double crossover_random = crossover_dist(rand_gen);
	if(crossover_random > crossover_rate)
		return;

	// TODO: verify if points valid
	thrust::uniform_int_distribution<unsigned> start_dist(0, member_size - 1);
	rand_gen.discard(idx);
	unsigned start = start_dist(rand_gen);
	thrust::uniform_int_distribution<unsigned> size_dist(0,
			member_size - 1 - start);
	rand_gen.discard(idx);
	unsigned swap_size = size_dist(rand_gen);
	uint8_t* temp_buffer = new uint8_t[swap_size];

	memcpy(temp_buffer, parent1 + start, swap_size);
	memcpy(parent1 + start, parent2 + start, swap_size);
	memcpy(parent2 + start, temp_buffer, swap_size);
	delete[] temp_buffer;
}

__device__ static void crossover(int idx, uint8_t* selected,
	thrust::default_random_engine& rand_gen) {
	if(idx % 2 == 0) {
		two_point_crossover(idx, get_member(selected, idx),
			get_member(selected, idx+1), rand_gen);
	}
	#if 0
	else {
		printf("thread number %d doing nothing in crossover\n", idx);
	}
	#endif
}

// TODO: consider speeding up mutation by not waiting to sync threads after crossover
// TODO: consider speeding up by having thread per bit and mutating
__device__ static void mutation(int idx, uint8_t* crossed_over,
	thrust::default_random_engine& rand_gen) {
	thrust::uniform_real_distribution<double> mutation_dist(0.0, 1.0);
	uint8_t* member = get_member(crossed_over, idx);
	for(unsigned i = 0; i < member_size; i++) {
		rand_gen.discard(idx);
		double mutation_prob = mutation_dist(rand_gen);
		if(mutation_prob > mutation_rate)
			continue;
		assert(member[i] == 0 || member[i] == 1);
		// flip the 'bit' when mutating
		member[i] ^= 1;
	}
}

__device__ void swap_population(uint8_t** population_1, uint8_t** population_2) {
	uint8_t* temp = *population_1;
	*population_1 = *population_2;
	*population_2 = temp;
}

// I think it's safe to assume that pop_size <= 1024
// so just one block with many threads
// TODO: initalize population
__global__ void genetic_kernel() {
	int idx = threadIdx.x;
	//printf("idx is %d\n", idx);
	__shared__ uint8_t* population;
	__shared__ uint8_t* new_population;
	__shared__ double* fitness_arr;
	__shared__ double* best_fitnesses;
	__shared__ double best_fit;
	thrust::default_random_engine rand_gen;
	

	//__shared__ hiprandState_t* rand_state;
	if(idx == 0) {
		population = new uint8_t[pop_size * member_size];
		new_population = new uint8_t[pop_size * member_size];
		fitness_arr = new double[pop_size];
		best_fitnesses = new double[pop_size];
	}
	__syncthreads();
	best_fitnesses[idx] = -DBL_MAX;
	init_population(idx, population, rand_gen);
	
	#if 0
	if(idx == 0)
		print_pop(population);
	__syncthreads();
	#endif

	for(unsigned i = 0; i < max_iterations; i++) {
		// selection
		__syncthreads();
		selection(idx, population, fitness_arr, new_population, rand_gen, best_fitnesses);

		#if 0
		if(idx == 0) {
			__syncthreads();
			printf("----------------------------------------------\n");
			print_pop(new_population);
		}
		#endif

		__syncthreads();
		crossover(idx, new_population, rand_gen);

		__syncthreads();
		mutation(idx, new_population, rand_gen);
		
		__syncthreads();
		if(idx == 0)
			swap_population(&population, &new_population);
	}
	__syncthreads();
	parallel_max(best_fitnesses, &best_fit, pop_size);
	__syncthreads();
	if(idx == 0) {
		printf("best fitness: %f\n", 1/best_fit);
	}
}

void run_genetic_cuda(size_t p_size, size_t m_size, double cr_rate,
		double m_rate, unsigned max_iter, double (*fitness_func)(uint8_t*)) {
		//double m_rate, unsigned max_iter) {
	hipMemcpyToSymbol(HIP_SYMBOL(pop_size), &p_size, sizeof(size_t));
	hipMemcpyToSymbol(HIP_SYMBOL(member_size), &m_size, sizeof(size_t));
	hipMemcpyToSymbol(HIP_SYMBOL(crossover_rate), &cr_rate, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(mutation_rate), &m_rate, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(max_iterations), &max_iter, sizeof(unsigned));
	hipMemcpyToSymbol(HIP_SYMBOL(fitness), &fitness_func, sizeof(fitness_func));
	hipDeviceSynchronize();
	check_cuda_error("memcpying arguments");

	// TODO: change numbers in angle brackets later
	genetic_kernel<<<1, p_size>>>();
	hipDeviceSynchronize();
	check_cuda_error("after running kernel");
	std::cout << "done with cuda genetic\n";
}
