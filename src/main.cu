#include "genetic_seq.h"
#include "genetic_cuda.h"
#include <cassert>
#include <iostream>
#include <fstream>
#include <cstring>
#include <stack>
#include "args_parser.h"
#include "vertex_cover.h"
#include <vector>

std::string input_file;
unsigned pop_size;
unsigned function_evals;
RunMode run_mode;

__device__ fitness_t dummy = d_vertex_cover_fitness;

int main(int argc, char**argv) {
	handle_args(argc, argv);
	unsigned num_vertices = load_graph(input_file);
	std::cout << "|V| = " << num_vertices << std::endl;
	//std::cout << "greedy solution is " << greedy_vertex_cover() << std::endl;
	/*run_genetic_seq(50, num_vertices, 0.6, 1/(double)num_vertices, 20000/50,
			&vertex_cover_fitness);*/

	fitness_t cuda_fitness;
	hipMemcpyFromSymbol(&cuda_fitness, HIP_SYMBOL(dummy), sizeof(fitness_t));
	hipError_t err1 = hipGetLastError();
	if(err1 != hipSuccess) {
		std::cerr << "couldn't copy func ptr\n";
		std::cerr << "\t" << hipGetErrorString(err1) << std::endl;
		return 1;
	}
	/*run_genetic_cuda(50, num_vertices, 0.6, 1/(double)num_vertices, 20000/50,
			cuda_fitness);*/

	std::cout << "running mode is " << run_mode << std::endl;

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	if(run_mode == CUDA) {
		run_genetic_cuda(pop_size, num_vertices, 0.6, 1/(double)num_vertices, function_evals/pop_size,
			cuda_fitness);
	}

	else if(run_mode == SEQUENTIAL) {
		run_genetic_seq(pop_size, num_vertices, 0.6, 1/(double)num_vertices, function_evals/pop_size,
			&vertex_cover_fitness);
	}
	
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	std::cout << "elapsed time: " << time << std::endl;

	destroy_graph();
}
